#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include "operaciones.h"



__global__ void multiplicaVectores(int* mres,int* m1,int* m2, int numFilas, int numColumnas)
{

	int idx=blockIdx.x*blockDim.x+threadIdx.x;

	int numFila=idx/numColumnas;
	int numColumna=idx%numColumnas;

	mres[numFila*numColumnas+numColumna]=0;
	for(int i=0;i<numColumnas;i++)
	{
		mres[numFila*numColumnas+numColumna]+=m1[numFila*numColumnas+i]*m2[numFila*numColumnas+i];
	}

}



void imprimeMatriz(int* mat, int numFilas, int numColumnas)
{
	for(int i=0;i<numFilas;i++)
	{
		for(int j=0;j<numColumnas;j++)
		{
			printf("%d,",mat[i*numFilas+j]);
		}
		printf("\n");
	}

}


int multiplicaVectores(int* v1,int* v2, int size)
{
	int resultado=0;
	for(int i=0;i<size;i++)
	{
		resultado+=v1[i]*v2[i];
	}
	return resultado;
}




void multiplicaMatrices(int cpu,int* m1,int* m2, int* mRes,int numFilasM1, int numColumnasM1,int numFilasM2, int numColumnasM2)
{

if(cpu){
printf("CPU!\n");
		for(int i=0;i<numFilasM1;i++)
			for(int j=0;j<numColumnasM2;j++)
			mRes[i*numColumnasM2+j]=multiplicaVectores(&(m1[i*numColumnasM2]),
						&(m2[j*numColumnasM2])
						,numColumnasM2);
}else{
printf("CUDA!\n");

	int numThreadBloque=128;
	int numBloques=(numFilasM1*numColumnasM2/numThreadBloque)+1;
	
	int* d_m1;
	int* d_m2;
	int* d_mres;

	hipMalloc((void**)&d_m1,sizeof(int)*numFilasM1*numColumnasM1);
	hipMalloc((void**)&d_m2,sizeof(int)*numFilasM2*numColumnasM2);
	hipMalloc((void**)&d_mres,sizeof(int)*numFilasM1*numColumnasM2);
	
	hipMemcpy(d_m1,m1,sizeof(int)*numFilasM1*numColumnasM1,hipMemcpyHostToDevice);
	hipMemcpy(d_m2,m2,sizeof(int)*numFilasM2*numColumnasM2,hipMemcpyHostToDevice);

multiplicaVectores<<<numThreadBloque,numBloques>>>(d_mres,d_m1,d_m2,numFilasM1, numColumnasM2);

	hipMemcpy(mRes,d_mres,sizeof(int)*numFilasM1*numColumnasM2,hipMemcpyDeviceToHost);
}
}
