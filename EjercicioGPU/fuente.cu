#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>


__global__ void sumaDatos(int* in, int* out, int size)
{
	int IDx=blockIdx.x*blockDim.x+threadIdx.x;

	if(IDx>size) return;

	out[IDx]=in[IDx]+in[IDx];

}

int main(int argc, char **argv)
{
	int datosCount=100000000;

	int* h_datos=(int*)malloc(datosCount*sizeof(int));
	int* h_datosout=(int*)malloc(datosCount*sizeof(int));
	int* d_datos;
	int* d_datosout;
	hipMalloc(&d_datos,datosCount*sizeof(int));
	hipMalloc(&d_datosout,datosCount*sizeof(int));


	for(int i=0;i<datosCount;i++)
	{
		h_datos[i]=i*2;
	}

	hipMemcpy(d_datos,h_datos,datosCount*sizeof(int),hipMemcpyHostToDevice);
	int numthreads=256;
	int numbloques=datosCount/numthreads+1;

	sumaDatos<<<numbloques,numthreads>>>(d_datos,d_datosout,datosCount);

	hipMemcpy(h_datosout,d_datosout,datosCount*sizeof(int),hipMemcpyDeviceToHost);

	printf("FIN\n");

	return 0;
}
